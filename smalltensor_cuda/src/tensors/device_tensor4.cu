#ifndef device_TENSOR4_HPP_
#define device_TENSOR4_HPP_
#include "../utils/__utils.h"
#include "../expressions/expressions.h"

using namespace std;
template <typename _d_dat_t, std::size_t _d_d1, std::size_t _d_d2, std::size_t _d_d3, std::size_t _d_d4>
class device_tensor4
{
public:
	_d_dat_t* __restrict__ d_data_;

	device_tensor4()
	// : d_data_{new _d_dat_t[_d_d1*_d_d2]}
	{

	}
	device_tensor4(device_tensor4 const& d_rhs_)
	// : d_data_{new _d_dat_t[_d_d1*_d_d2]}
	{
		hipMalloc((void**) &d_data_, sizeof(_d_dat_t)*_d_d1*_d_d2*_d_d3*_d_d4);
		DEBUG_MSG("device_tensor4 copy constructor is called");
		// std::memcpy(d_data_, d_rhs_.d_data_, sizeof(d_data_)*_d_d1*_d_d2*_d_d3*_d_d4);
		hipMemcpy(d_data_, d_rhs_.d_data_ , sizeof(d_data_)*_d_d1*_d_d2*_d_d3*_d_d4, hipMemcpyDeviceToDevice);
	}
	device_tensor4& operator=(device_tensor4 const& d_rhs_){

		DEBUG_MSG("device_tensor4 copy assignment operator is called");
		if(this != &d_rhs_){
			hipMemcpy(d_data_, d_rhs_.d_data_ , sizeof(d_data_)*_d_d1*_d_d2*_d_d3*_d_d4, hipMemcpyDeviceToDevice);
		}
		return *this;
	}

    device_tensor4(device_tensor4&& d_rhs_) noexcept
    // : d_data_{new _d_dat_t[_d_d1*_d_d2*_d_d3*_d_d4]}
    {
    	hipMalloc((void**) &d_data_, sizeof(_d_dat_t)*_d_d1*_d_d2*_d_d3*_d_d4);
    	std::swap(d_data_, d_rhs_.d_data_);
    	DEBUG_MSG("device_tensor4 move constructor is called");
    }
    device_tensor4& operator=(device_tensor4&& d_rhs_) noexcept{
    	if(this != &d_rhs_){
    		std::swap(d_data_, d_rhs_.d_data_);
    	}
    	DEBUG_MSG("device_tensor4 move assignment operator is called");
    	return *this;
    }
	~device_tensor4(){
		if (d_data_!=nullptr){
		    hipFree(d_data_);
		    d_data_=nullptr;
		}
	}
	// device_tensor4(std::string const& other_):d_data_{new _d_dat_t[_d_d1*_d_d2*_d_d3]}{
	//     if (other_ == "identity")
	//     {
	//         ASSERT_MSG(_d_d1==_d_d2*_d_d3, "ERROR:device_tensor4 has different dimensions, cannot be identity.");
	//         for (std::size_t n1 = 0; n1 < _d_d1; ++n1)
	//         {
	//             (*this)(n1,n1) = 1;
	//         }
	//     }
	// }
	// inline _d_dat_t& operator()(std::size_t n1_, std::size_t n2_){
	// 	ASSERT_MSG(n1_< _d_d1 && n2_ < _d_d2*_d_d3, "device_tensor4() index out of bounds in lvalue. ");
	// 	return d_data_[ n1_ * _d_d2*_d_d3 + n2_];
	// }
	// inline _d_dat_t operator()(std::size_t n1_, std::size_t n2_)const{
	// 	ASSERT_MSG(n1_< _d_d1 && n2_ < _d_d2*_d_d3, "device_tensor4() index out of bounds in rvalue. ");
	// 	return d_data_[ n1_ * _d_d2*_d_d3 + n2_];
	// }
	// template <char i, char j>
	// inline expr2<_d_dat_t, _d_d1, _d_d2*_d_d3, i, j>& operator()(Index<i> i_, Index<j> j_){
 //        return static_cast<expr2<_d_dat_t, _d_d1, _d_d2*_d_d3, i, j>&>(*this);
	// }

	// template <char i, char j>
	// inline expr2<_d_dat_t, _d_d1, _d_d2, i, j> const& operator()(Index<i> i_, Index<j> j_)const{
 //        return static_cast<expr2<_d_dat_t, _d_d1, _d_d2, i, j>const&>(*this);
	// }

	// template <char i>
	// inline _d_dat_t operator()(Index<i> i_, Index<i/*same*/> j_){
	// 	ASSERT_MSG(_d_d1 == _d_d2, "Dimension size should be equal for dummy indices. ");
	// 	_d_dat_t ret=0;
	// 	for (std::size_t n1 = 0; n1 < _d_d1; ++n1){
	// 		ret += (*this)(n1,n1);
	// 	}
 //        return ret;
	// }

	// inline device_tensor4& operator*=(_d_dat_t const& scalar_){
	// 	for (std::size_t n1 = 0; n1 < _d_d1; ++n1){
	// 		for (std::size_t n2 = 0; n2 < _d_d2; ++n2){
	// 			(*this)(n1,n2) *= scalar_ ;
	// 		}
	// 	}
	// 	return (*this);
	// }
};

#endif
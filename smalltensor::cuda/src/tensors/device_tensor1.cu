#include "hip/hip_runtime.h"
#pragma once
#include "../utils/__utils.h"
// #include "../expressions/expressions.h"

using namespace std;
template <typename _d_dat_t, std::size_t _d_d1>
class device_tensor1
{
public:
	_d_dat_t* __restrict__ d_data_;

	device_tensor1()
	// : d_data_{new _d_dat_t[_d_d1]}
	{
		// hipMalloc((void**) &(this.d_data_), sizeof(_d_dat_t)*_d_d1);
	}
	device_tensor1(device_tensor1 const& d_rhs_)
	// : d_data_{new _d_dat_t[_d_d1]}
	{
		hipMalloc((void**) &d_data_, sizeof(_d_dat_t)*_d_d1);
		DEBUG_MSG("device_tensor1 copy constructor is called");
		// std::memcpy(d_data_, rhs_.d_data_, sizeof(d_data_)*_d_d1);
		hipMemcpy(d_data_, d_rhs_.d_data_ , sizeof(d_data_)*_d_d1, hipMemcpyDeviceToDevice);
	}
	device_tensor1& operator=(device_tensor1 const& d_rhs_){
		DEBUG_MSG("device_tensor1 copy assignment operator is called");
		if(this != &d_rhs_){
	        // std::memcpy(d_data_, rhs_.d_data_, sizeof(d_data_)*_d_d1);		
			hipMemcpy(d_data_, d_rhs_.d_data_ , sizeof(d_data_)*_d_d1, hipMemcpyDeviceToDevice);
		}
		return *this;
	}
    device_tensor1(device_tensor1&& d_rhs_) noexcept    
    // : d_data_{new _d_dat_t[_d_d1]}
    {
		hipMalloc((void**) &d_data_, sizeof(_d_dat_t)*_d_d1);
    	std::swap(d_data_, d_rhs_.d_data_);
    	DEBUG_MSG("device_tensor1 move constructor is called");

    }
    device_tensor1& operator=(device_tensor1&& d_rhs_) noexcept{
    	DEBUG_MSG("device_tensor1 move assignment operator is called");
    	if(this != &d_rhs_){
    		std::swap(d_data_, d_rhs_.d_data_);
    	}
    	return *this;
    }
	~device_tensor1(){
		if (d_data_!=nullptr){
		    hipFree(d_data_);
		    d_data_=nullptr;
		}
	}

	// __global__ inline _d_dat_t& operator()(std::size_t n1_){
	// 	ASSERT_MSG(n1_< _d_d1 , "device_tensor1() index out of bounds in lvalue. ");
	// 	return d_data_[ n1_ ];
	// }
	// __global__ inline _d_dat_t operator()(std::size_t n1_)const{
	// 	ASSERT_MSG(n1_< _d_d1 , "device_tensor1() index out of bounds in rvalue. ");
	// 	return d_data_[ n1_ ];
	// }
	// template <char i>
	// inline expr1<_d_dat_t, _d_d1, i>& operator()(Index<i> i_){
 //        return static_cast<expr1<_d_dat_t, _d_d1, i>&>(*this);
	// }
	// template <char i>
	// inline expr1<_d_dat_t, _d_d1, i> const& operator()(Index<i> i_)const{
 //        return static_cast<expr1<_d_dat_t, _d_d1, i>const&>(*this);
	// }
	// inline device_tensor1& operator*=(_d_dat_t const& scalar_){
	// 	for (std::size_t n1 = 0; n1 < _d_d1; ++n1){
	// 		(*this)(n1) *= scalar_ ;
	// 	}
	// 	return (*this);
	// }

};


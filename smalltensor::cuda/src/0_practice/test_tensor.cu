#pragma once
#include <hip/hip_runtime.h>

// #include "../utils/__utils.h"
// #include "../expressions/expressions.h"
#include <iostream>
using namespace std;
// template <typename val_type>
template <typename val_type, std::size_t dim>
class device_tensor1
{
public:
	val_type* d_data_;

	device_tensor1()
	// : d_data_{new val_type[dim]}
	{
		// cudaMalloc((void**) &(this.d_data_), sizeof(val_type)*dim);
	}
	device_tensor1(device_tensor1 const& d_rhs_)
	// : d_data_{new val_type[dim]}
	{
		hipMalloc((void**) &d_data_, sizeof(val_type)*dim);
		// DEBUG_MSG("device_tensor1 copy constructor is called");
		// std::memcpy(d_data_, rhs_.d_data_, sizeof(d_data_)*dim);
		hipMemcpy(d_data_, d_rhs_.d_data_ , dim, hipMemcpyDeviceToDevice);
	}
	device_tensor1& operator=(device_tensor1 const& d_rhs_){
		// DEBUG_MSG("device_tensor1 copy assignment operator is called");
		if(this != &d_rhs_){
	        // std::memcpy(d_data_, rhs_.d_data_, sizeof(d_data_)*dim);		
			hipMemcpy(d_data_, d_rhs_.d_data_ , dim, hipMemcpyDeviceToDevice);
		}
		return *this;
	}
    device_tensor1(device_tensor1&& d_rhs_) noexcept    
    // : d_data_{new val_type[dim]}
    {
		hipMalloc((void**) &d_data_, sizeof(val_type)*dim);
    	std::swap(d_data_, d_rhs_.d_data_);
    	// DEBUG_MSG("device_tensor1 move constructor is called");

    }
    device_tensor1& operator=(device_tensor1&& d_rhs_) noexcept{
    	// DEBUG_MSG("device_tensor1 move assignment operator is called");
    	if(this != &d_rhs_){
    		std::swap(d_data_, d_rhs_.d_data_);
    	}
    	return *this;
    }
	~device_tensor1(){
		if (d_data_!=nullptr){
		    hipFree(d_data_);
		    d_data_=nullptr;
		}
	}


};

int main(int argc, char const *argv[])
{
	device_tensor1<float,3> test ;
	/* code */
	return 0;
}